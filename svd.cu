#include "hip/hip_runtime.h"
#include <string>
#include <numeric>
#include <vector>
#include <iostream>
#include <iterator>
#include <sstream>
#include <algorithm>
#include <fstream>
#include <stdio.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <assert.h>
#include <functional>
#include <Eigen/Sparse>
#include <chrono>
#include <omp.h>

#include "svd_GPU.hpp"

using namespace std;
using namespace Eigen;
using namespace std::chrono;

// generator:
struct c_unique {
  int current;
  c_unique() {current=0;}
  int operator()() {return current++;}
} UniqueNumber;

void SVD::set_values (int k, double et, double r, double ep,
                      double max_ep) {
    M = NUM_USERS_SMALL;   
    N = NUM_MOVIES;
    K = k;
    eta = et;
    reg = r;
    eps = ep;
    max_epochs = max_ep;

    cout << "done setting values\n";
}


void SVD::initialize () {

    // Allocate device memory for the matrices
    CUDA_CALL(hipMalloc((void **) &u_mat, 
                    NUM_USERS_SMALL * K * sizeof(double)));

    CUDA_CALL(hipMalloc((void **) &v_mat, 
                    NUM_MOVIES * K * sizeof(double)));

    // Convert U and V into flat arrays in a gross way

    float * u_flat = new float[NUM_USERS_SMALL * K];
    for (i = 0; i < NUM_USERS_SMALL; i++) {
        copy(U[i].begin(), U[i].end(), u_flat);
        u_flat += U[i].size();
    }    

    float * v_flat = new float[NUM_MOVIES * K];
    for (i = 0; i < NUM_MOVIES; i++) {
        copy(V[i].begin(), V[i].end(), v_flat);
        v_flat += V[i].size();
    }    

    // Copy over the data onto the device
    CUBLAS_CALL(hipblasSetMatrix(NUM_USERS_SMALL, K, sizeof(double), 
                        U, NUM_USERS_SMALL, u_mat, NUM_USERS_SMALL));
    
    CUBLAS_CALL(hipblasSetMatrix(NUM_MOVIES, K, sizeof(double), 
                        v_flat, NUM_MOVIES, v_mat, NUM_MOVIES));   

    // TODO: do we need to transpose things for row/col major purposes???
    // the default is col major but we have our data in row major??? 

    cout << "done initializing GPU stuff \n";
}

void SVD::load_data() {

    cout << "load training data \n";

    ifstream file("Archive/small_train.txt");

    // stupid way of initializing for now
    for (int i = 0; i < NUM_USERS_SMALL; ++i)
    {
        vector<double> temp(NUM_MOVIES, 0);
        Y.push_back(temp);
    }

    cout << "made it here \n";
    int uid, mid, date, rating;
    while (file >> uid >> mid >> date >> rating) {
        Y[uid-1][mid-1] = rating;
    }

    cout << "made it here 2\n";
}

// void SVD::load_valid() {
//     //  CROSS VALIDATING ON VALID AND HIDDEN (4% OF ALL TRAINING DATA)

//     cout << "load validation data\n";

//     //ifstream file("../data/Archive/tiny_qual.txt"); // actually qual.txt
//     ifstream file("Archive/small_probe.txt");

//     int uid, mid, date, rating;
//     while (file >> uid >> mid >> date >> rating) {
//         val_uid.push_back(uid-1);
//         val_mid.push_back(mid-1);
//         val_ratings.push_back(rating);
//     }
// }

double SVD::get_err() {
    auto start = high_resolution_clock::now();

    cout << "compute error\n";

    double squared_err = 0.0;

    for (auto row = Y.begin(); row != Y.end(); ++row) {
        
        for (auto col = row->begin(); col != row->end(); ++col) {
            int Y_ij = (int) *col; // actual rating value
            int i = distance(Y.begin(), row); // row index
            int j = distance(row->begin(), col); // col index

            cout << "got index\n";
           
            vector<double> urow = U[i];
            vector<double> vrow = V[i];

            // DONE: squared_err += 0.5 * pow((Y_ij - urow.dot(vrow)), 2);

            double * dev_urow;
            double * dev_vrow;

            // Convert from vector to c array
            double * urow_array = &urow[0];
            double * vrow_array = &vrow[0];

            // Allocate device memory for the vectors
            CUDA_CALL(hipMalloc((void **) &dev_urow, K * sizeof(double)));

            CUDA_CALL(hipMalloc((void **) &dev_vrow, K * sizeof(double)));

            // Compute the dot product of urow and vrow
            double * dev_dot;
            hipblasDdot(handle, K, dev_urow, 1, dev_vrow, 1, dev_dot);

            // Copy the result from device to host machine
            double dot;
            hipMemcpy(&dot, void *dev_dot, sizeof(double), hipMemcpyDeviceToHost);

            squared_err += 0.5 * pow((Y_ij - dot), 2);

            // Free the cuda memory 
            // TODO: do we need to use hipblasDestroy????
            hipFree(dev_urow);
            hipFree(dev_vrow);
        }
    }

    squared_err = squared_err / Y.size();

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "computed error: " << squared_err << " ; time taken: "<<duration.count()<<endl;;

    return squared_err;
}

vector<double> SVD::grad_U(vector<double> U_i, vector<double> V_j, int Y_ij) {
    cout << "begin grad U\n";

    // DONE: convert to cuBLAS
    // double prod = U_i.adjoint() * V_j;
    // double Y_ij_d = (double) Y_ij;
    // VectorXd sq_err = V_j * (Y_ij_d - prod);
    // VectorXd gradient = (U_i * reg - sq_err) * eta;

    double * dev_urow;
    double * dev_vrow;

    // convert from vector to c array
    double * urow_array = &U_i[0];
    double * vrow_array = &V_i[0];

    // Allocate device memory for the vectors
    CUDA_CALL(hipMalloc((void **) &dev_urow, K * sizeof(double)));

    CUDA_CALL(hipMalloc((void **) &dev_vrow, K * sizeof(double)));  

    // Copy arrays from the host to the device
    hipMemcpy(dev_urow, urow_array, sizeof(double) * K, hipMemcpyHostToDevice);
    hipMemcpy(dev_vrow, vrow_array, sizeof(double) * K, hipMemcpyHostToDevice);

    // Compute the dot product of urow and vrow
    double * dev_dot;
    hipblasDdot(handle, K, dev_urow, 1, dev_vrow, 1, dev_dot);  

    // Copy the result from device to host machine
    double dot;
    hipMemcpy(&dot, void *dev_dot, sizeof(double), hipMemcpyDeviceToHost);

    double Y_ij_d = (double) Y_ij;

    // sq_err = V_j * (Y_ij_d - prod)
    double alpha = -1 * Y_ij_d - dot;
    hipblasDscal(handle, K, &alpha, dev_vrow, 1); // dev_vrow now contains -sq_err

    // gradient = (U_i * reg - sq_err) * eta;

    // dev_urow now contains U_i * reg
    hipblasDscal(handle, K, &reg, dev_urow, 1); 

    // dev_vrow now contains (U_i * reg) - sq_err
    hipblasDaxpy(handle, K, 1.0, dev_urow, 1, dev_vrow, 1);

    // dev_vrow now contains (U_i * reg - sq_err) * eta
    hipblasDscal(handle, K, &eta, dev_vrow, 1); 

    double * gradient_array;
    // Copy over the data from the device
    hipMemcpy(gradient_array, void *dev_vrow, sizeof(double) * K, hipMemcpyDeviceToHost);

    // convert c array to vector
    vector<double> gradient (gradient_array, 
            gradient_array + sizeof gradient_array / sizeof gradient_array[0]);

    // Free the cuda memory
    hipFree(dev_urow);
    hipFree(dev_vrow);
    hipFree(dev_dot);

    return gradient;
}

vector<double> SVD::grad_V(vector<double> U_i, vector<double> V_j, int Y_ij) {
    cout << "begin grad V\n";

    // DONE: convert to cuBLAS
    // double prod = U_i.adjoint() * V_j;
    // double Y_ij_d = (double) Y_ij;
    // VectorXd sq_err = U_i * (Y_ij_d - prod);
    // VectorXd gradient = (V_j * reg - sq_err) * eta;

    double * dev_urow;
    double * dev_vrow;

    // convert from vector to c array
    double * urow_array = &U_i[0];
    double * vrow_array = &V_i[0];

    // Allocate device memory for the vectors
    CUDA_CALL(hipMalloc((void **) &dev_urow, K * sizeof(double)));

    CUDA_CALL(hipMalloc((void **) &dev_vrow, K * sizeof(double)));  

    // Copy arrays from the host to the device
    hipMemcpy(dev_urow, urow_array, sizeof(double) * K, hipMemcpyHostToDevice);
    hipMemcpy(dev_vrow, vrow_array, sizeof(double) * K, hipMemcpyHostToDevice);

    // Compute the dot product of urow and vrow
    double * dev_dot;
    hipblasDdot(handle, K, dev_urow, 1, dev_vrow, 1, dev_dot);  

    // Copy the result from device to host machine
    double dot;
    hipMemcpy(&dot, void *dev_dot, sizeof(double), hipMemcpyDeviceToHost);

    double Y_ij_d = (double) Y_ij;

    // sq_err = U_i * (Y_ij_d - prod)
    double alpha = -1 * Y_ij_d - dot;
    hipblasDscal(handle, K, &alpha, dev_urow, 1); // dev_urow now contains -sq_err

    // gradient = (V_j * reg - sq_err) * eta;

    // dev_vrow now contains V_j * reg
    hipblasDscal(handle, K, &reg, dev_vrow, 1); 

    // dev_urow now contains V_j * reg - sq_err
    hipblasDaxpy(handle, K, 1.0, dev_vrow, 1, dev_urow, 1);

    // dev_urow now contains (V_j * reg - sq_errU_i * reg - sq_err) * eta
    hipblasDscal(handle, K, &eta, dev_urow, 1); 

    double * gradient_array;
    // Copy over the data from the device
    hipMemcpy(gradient_array, void *deu_vrow, sizeof(double) * K, hipMemcpyDeviceToHost);

    // convert c array to vector
    vector<double> gradient (gradient_array, 
            gradient_array + sizeof gradient_array / sizeof gradient_array[0]);

    // Free the cuda memory
    hipFree(dev_urow);
    hipFree(dev_vrow);
    hipFree(dev_dot);

    return gradient;
}

// Take in two vectors of doubles, convert to arrays in host memory, and 
// perform a - b. Copy array to host memory and return in vector form.

void SVD::sub_vectors(vector<double> a, vector<double> b) {

    double * dev_a;
    double * dev_b;

    // multiply b by -1 element-wise
    for (int i = 0; i < b.size(); ++i)
    {
        b[i] = b[i] * -1;
    }

    // convert from vector to c array
    double * a_array = &a[0];
    double * b_array = &b[0];

    // Copy arrays from the host to the device
    hipMemcpy(dev_a, a_array, sizeof(double) * K, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b_array, sizeof(double) * K, hipMemcpyHostToDevice);

    hipblasDaxpy(handle, K, 1.0, dev_a, 1, dev_b, 1); // result in dev_b

    double * sub_array;
    // Copy over the data from the device
    hipMemcpy(sub_array, void *dev_b, sizeof(double) * K, hipMemcpyDeviceToHost);

    // convert c array to vector
    vector<double> sub (sub_array, 
            sub_array + sizeof sub_array / sizeof sub_array[0]);

    // Free the cuda memory
    hipFree(dev_urow);
    hipFree(dev_vrow);

    return sub;
}

void SVD::train_model() {
    // U = M x K, V = N x K
    // rating Y_ij is approximatedby (UV^T)_ij

    cout << "begin training\n";

    // fill feature matrices with random 0-0.5 values
    // double lower_bound = -0.5;
    // double upper_bound = 0.5;
    // uniform_real_distribution<double> unif(lower_bound, upper_bound);
    // default_random_engine re;
    // generate(U.begin(), U.end(), unif(re));
    // generate(V.begin(), V.end(), unif(re));

    // for now, just 0
    // TODO: fix, this is really stupid
    for (int i = 0; i < NUM_USERS_SMALL; ++i)
    {
        vector<double> temp;
        for (int j = 0; i < K; ++j)
        {
            temp.push_back(0.0);
        }
        U.push_back(temp);
    }

    for (int i = 0; i < NUM_MOVIES; ++i)
    {
        vector<double> temp;
        for (int j = 0; i < K; ++j)
        {
            temp.push_back(0.0);
        }
        V.push_back(temp);
    }

    // get initial error
    double err0 = get_err();
    double err = err0;
    double err1 = 0.0;

    vector<int> indices(NUM_USERS_SMALL);
    indices.reserve(NUM_USERS_SMALL);

    generate (indices.begin(), indices.end(), UniqueNumber);

    // continue for max_epochs
    for (int e=0; e<max_epochs; ++e) {
        cout << "begin epoch" << e << "\n";

        auto start = high_resolution_clock::now();

        // shuffle the points in the indices vector
        random_shuffle(indices.begin(), indices.end());

        vector<vector<double>> Y_perm = Y;
        cout << "completed permutation\n";


        // update U and V
        for (vector<int>::iterator it=indices.begin(); it!=indices.end(); ++it) {
            int k = *it;

            // potential problem: no easy way to only iterate through the 
            // non zero values like there was for eigen so it might be slower

            for (auto row = Y.begin(); row != Y.end(); ++row)
            {
                for (auto col = row->begin(); col != row->end(); ++col) {
                    int Y_ij = (int) *col; // actual rating value
                    int i = distance(Y.begin(), row); // row index
                    int j = distance(row->begin(), col); // col index

                    // Update U
                    vector<double> gradu = grad_U(U[i], V[j], Y_ij);
                    vector<double> urow = U[i];
                    // DONE: Convert U[i] = urow - gradu to CuBLAS
                    U[i] = sub_vectors(urow, gradu);

                    cout << "completed grad U\n";

                    // Update V
                    vector<double> gradv = grad_V(U[i], V[j], Y_ij);
                    vector<double> vrow = V[i];
                    // DONE: Convert V[i] = vrow - gradv to CuBLAS
                    V[i] = sub_vectors(vrow, gradv);

                    cout << "completed grad V\n";
                }
            }            

        cout << "updated U, V for all points in training set\n";

        double err_prev = err;
        err = get_err();

        if (e == 0) {
            err1 = err;
        }

        // check if error reduction satisfied
        double comp = (err_prev - err) / (err0 - err1);
        cout << "error reduction = " << comp << "\n";
        if (comp <= eps) {
            cout << " YEET\n";
            break;
        }

        auto stop = high_resolution_clock::now();
        auto duration = duration_cast<microseconds>(stop - start);
        cout << "computed updated error for epoch " << e << "; total time taken: "<<duration.count()<<"\n";
    }

    // set final e_in
    double e_in = err;

    cout << "completed training, e_in = " << e_in << "\n";
    }
}

void SVD::predict_valid() {
    // cout << "predict probe\n";

    // double squared_err = 0.0;

    // for (int i=0; i<val_uid.size(); ++i) {
    //     int uid = val_uid.at(i);
    //     int mid = val_mid.at(i);
    //     double rating = (double) val_ratings.at(i);

    //     VectorXd urow = U.row(uid);
    //     VectorXd vrow = V.row(mid);

    //     double prediction = urow.adjoint() * vrow;

    //     val_predictions.push_back(rating);

    //     squared_err += 0.5 * pow((rating - prediction), 2);
    // }

    // squared_err = squared_err / val_uid.size();

    // cout << "completed predictions for probe set, e_out = " << squared_err << "\n";

    // // yeet it out to a file
    // ofstream of("svd_val_results.txt");
    // ostream_iterator<double> output_iterator(of, "\n");
    // copy(val_predictions.begin(), val_predictions.end(), output_iterator);
}

int main () {
    SVD svd;

    cout << "begin SVD...\n";

    // Initialize handle
    CUBLAS_CALL(hipblasCreate(&handle));

    auto start = high_resolution_clock::now();
    svd.load_data();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop-start);
    cout << "loading data took: " << duration.count() << endl;

    svd.set_values(50, 0.05, 0.01, 0.0001, 100);

    start = high_resolution_clock::now();
    svd.train_model();
    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop-start);
    cout << "training model took: " << duration.count() << endl;

    // svd.load_valid();

    // start = high_resolution_clock::now();
    // svd.predict_valid();
    // stop = high_resolution_clock::now();
    // duration = duration_cast<microseconds>(stop-start);
    // cout << "predicting valid took: " << duration.count() << endl;

    return 0;
}