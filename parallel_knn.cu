#include "hip/hip_runtime.h"
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "knn.cuh"
//#include "gpu_data.h"
#define gpu_errchk(ans) { gpu_assert((ans), __FILE__, __LINE__); }
inline void gpu_assert(hipError_t code, const char *file, int line,
                       bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "gpu_assert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        exit(code);
    }
}
__device__
int
cudaGetPairedUserRatingsNumber(float* data_lib, int size_i, int cij_length, int tid) {
    int counter_i = 0;
    int counter_j = 0;
    int res_counter = 0;
    while(counter_i < size_i and counter_j < size_i)
    {
        printf("id: %d jid: %d \n", data_lib[tid*size_i + counter_i*3], data_lib[cij_length*size_i + counter_j*3]);
        printf("counter i: %d, counter j: %d \n", counter_i, counter_j);
        if (data_lib[tid*size_i + counter_i*3] <= 0)
        {
            counter_i ++;
        }
        else if(data_lib[cij_length*size_i + counter_j*3] <= 0)
        {
            counter_j ++;
        }
        else if ((data_lib[tid*size_i + counter_i*3]) == (data_lib[cij_length*size_i + counter_j*3]))
        {
            if ((data_lib[cij_length*size_i + counter_j*3 + 1])!= 0 and 0 != (data_lib[tid*size_i + 1+3*counter_i]))
            {
                res_counter ++;
            }
            counter_i ++;
            counter_j ++;
        }
        else {
            if ((data_lib[tid*size_i + 3*counter_i]) > (data_lib[cij_length*size_i + 3*counter_j]))
            {
                counter_j ++;
            }
            else {
                counter_i ++;
            }
        }
    }
    return res_counter;
}


__device__ void
cudaGetPairedUserRatings(float* movie_i, float* movie_j, int size_i, bool first, float* res) {
    int counter_i = 0;
    int counter_j = 0;
    int res_counter = 0;
    while(counter_i < size_i and counter_j < size_i)
    {
        //
        if (movie_i[counter_i*3] < 0)
        {
            counter_i ++;
        }
        else if(movie_j[counter_j*3] < 0)
        {
            counter_j ++;
        }
        else if ((movie_i[counter_i*3]) == (movie_j[counter_j*3]))
        {
            if ((movie_j[counter_j*3 + 1])!= 0 and 0 != (movie_i[1+3*counter_i]))
            {
                if (first)
                {
                    res[res_counter] = (movie_i[1+3*counter_i]);
                    res_counter ++;
                }
                else{
                    res[res_counter] = (movie_j[1+3*counter_j]);
                    res_counter ++;
                    
                }
            }
            counter_i ++;
            counter_j ++;
        }
        else {
            if ((movie_i[3*counter_i]) > (movie_j[3*counter_j]))
            {
                counter_j ++;
            }
            else {
                counter_i ++;
            }
        }
    }
}
__device__
float pearson(float* item_rats_i, float* item_rats_j, int item_rats_size, float *item_i_diff, float *item_j_diff)
{
    float L;
    float top = 0, bottom = 0;
    //hipMalloc((void **) &item_j_diff, item_rats_size*sizeof(float));
    float i_sum = 0, j_sum = 0;
    L = item_rats_size;
    if (L <= 1)
    {
        return 0;
    }
    for (int i = 0; i < L; i++)
    {
        i_sum += item_rats_i[i];
        j_sum += item_rats_j[i];
    }
    float x_i_mean = i_sum / L;
    float x_j_mean = j_sum / L;
    float MSE_i = 0;
    float MSE_j = 0;
    
    for(int i = 0; i < L; i++)
    {
        item_i_diff[i] = item_rats_i[i] - x_i_mean;
        item_j_diff[i] = item_rats_j[i] - x_j_mean;
        MSE_i += pow(item_i_diff[i], 2);
        MSE_j += pow(item_j_diff[i], 2);
    }
    for (int i = 0; i < L; i++)
    {
        top += item_i_diff[i]*item_j_diff[i];
    }
    top *= 1/(L-1);
    bottom = sqrt(1/(L-1) * MSE_i)*sqrt(1/(L-1) * MSE_j);
    
    if (bottom == 0)
    {
        return 0;
    }
    delete [] item_j_diff;
    delete [] item_i_diff;
    //hipFree(item_i_diff);
    //hipFree(item_j_diff);
    return top/bottom;
}

__device__
float get_cij(float* item_rats_i, float* item_rats_j, int alpha, int item_rats_size, float *item_diff_i, float * item_diff_j)
{
    return pearson(item_rats_i, item_rats_j, item_rats_size, item_diff_i, item_diff_j)*item_rats_size/(item_rats_size+alpha);
}


__global__
void cuda_get_cij_kernel(int user_id, int cur_movie_id, int gpu_data_max_size, float * gpu_data_lib, float * user_list_movies,
                         float *gpu_out_cij, int cij_length, float *cij_lib, int alpha)
{
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    while (thread_index < cij_length) {
        int id = user_list_movies[3*thread_index];
        if (id < cur_movie_id)
        {
            int temp = id; id = cur_movie_id; cur_movie_id = temp;
        }
        if (cij_lib[cur_movie_id*17770 + id] <= 0)
        {
            float* movie_i = (float*)malloc(gpu_data_max_size * sizeof(float));
            float* movie_j = (float*)malloc(gpu_data_max_size * sizeof(float));
            
            float *item_j_diff = (float * )malloc(gpu_data_max_size * sizeof(float));
            float *item_i_diff = (float*)malloc(gpu_data_max_size * sizeof(float));
            
            if (movie_i == NULL) {printf(" rip m i");} else {printf("yay moive_i ");}
            if (movie_j == NULL) {printf(" rip movie j ");}else {printf("yay moive_j ");}
            if (item_i_diff == NULL) {printf(" rip item_i_diff ");}else {printf("yay item_i ");}
            if (item_j_diff == NULL) {printf(" rip item_J_diff ");}else {printf("yay itme_j ");}
            for (int s = 0; s < 1; s++)
            {
                movie_i[s] = gpu_data_lib[cij_length * gpu_data_max_size + s];
                movie_j[s] = gpu_data_lib[thread_index * gpu_data_max_size + s];
            }
            
            memcpy((void *) &movie_i,  (void *) &gpu_data_lib[cij_length * gpu_data_max_size], gpu_data_max_size * sizeof(float));
            memcpy((void *) &movie_j,  (void *) &gpu_data_lib[thread_index * gpu_data_max_size], gpu_data_max_size * sizeof(float));
            
            int cij_list_size = cudaGetPairedUserRatingsNumber(gpu_data_lib, gpu_data_max_size, cij_length, thread_index);
            
            float* movie_rat_i = new float[cij_list_size];
            float* movie_rat_j = new float[cij_list_size];
            
            movie_rat_i = malloc(cij_list_size*sizeof(float));
            movie_rat_j = malloc(cij_list_size*sizeof(float));
            
            cudaGetPairedUserRatings(movie_i, movie_j, gpu_data_max_size, true, movie_rat_i);
            cudaGetPairedUserRatings(movie_i, movie_j, gpu_data_max_size, false, movie_rat_j);
            cij_lib[cur_movie_id*17770 + id] = cij_list_size;//get_cij(movie_rat_i, movie_rat_j, alpha, cij_list_size, item_i_diff, item_j_diff);
            delete [] movie_rat_i;
            delete [] movie_rat_j;
            
            delete [] movie_i;
            free(movie_i);
            free(movie_j);
            free(item_j_diff);
            free(item_i_diff);
        }
        gpu_out_cij[thread_index] = 1;//cij_lib[cur_movie_id*17770 +id];
        thread_index += blockDim.x * gridDim.x;
    }
}

void cudaCallCij(const unsigned int blocks,
                 const unsigned int threadsPerBlock, int user_id, int cur_movie_id, int gpu_data_max_size, float * gpu_data_lib, float * user_list_movies, float *gpu_out_cij, int cij_length, float *cij_lib, int alpha) {
    
    cuda_get_cij_kernel<<<blocks, threadsPerBlock>>>(user_id, cur_movie_id, gpu_data_max_size, gpu_data_lib, user_list_movies, gpu_out_cij, cij_length, cij_lib, alpha);
}
